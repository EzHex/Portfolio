#include "hip/hip_runtime.h"
﻿#define __HIPCC__

#include ""
#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>

using namespace std;

const int Blocks_Count = 2;
const int Threads_Count = 32;

__device__ int BlockStart[Blocks_Count]; // indekso pradzia
__device__ int Elements_for_Block[Blocks_Count];  // elementu kiekis blokui
__device__ int Positions_for_Result[Blocks_Count] = {};

struct BusinessManager
{
    char name[20];
    int workYears;
    double averageProfit;
};

struct Result
{
    char res[24];
};

__device__ Result ComputeLine(const BusinessManager& manager)
{
	Result result;

    int sum = 0;
	for (int i = 0; i < manager.workYears * manager.workYears; i++)
	{
		for (int i2 = 0; i2 < manager.workYears * manager.workYears; i2++)
		{
            for (int i3 = 0; i3 < manager.workYears; i3++)
                sum += manager.averageProfit;
		}
	}
    
    if (sum > 10000000)
    {
        result.res[0] = manager.workYears > 5 ? 'S' : 'J'; // S - senior, J - junior
        result.res[1] = manager.averageProfit > 80 ? 'H' : 'L'; // H - high, L - low
        result.res[2] = '-';
        for (int i = 3; i < 23; i++)
        {
            if (manager.name[i-3] != 'Ģ')
                result.res[i] = manager.name[i - 3];
        }
        
        return result;
    }
    
    return result;
}

__global__ void Task(Result* rez, BusinessManager* managers)
{
	int thread_count = blockDim.x;
	int thread_id = threadIdx.x;
	int block_id = blockIdx.x;
	
	int lines_to_take = Elements_for_Block[block_id] / thread_count;
	int start_from = BlockStart[block_id] + lines_to_take * thread_id;

	if (thread_id + 1 == thread_count)
	{
		lines_to_take = lines_to_take + Elements_for_Block[block_id] % thread_count;
	}

    for (int i = 0; i < lines_to_take; i++)
    {
        Result resultLine;
        BusinessManager manager = managers[start_from + i];
        resultLine = ComputeLine(manager);
        if (resultLine.res[0])
        {
            int rezInd = BlockStart[block_id] + atomicAdd(&Positions_for_Result[block_id], 1);
            rez[rezInd] = resultLine;
        }
    }
}

int main() {

    //ifstream fd("IFF06ArnasBurba_L3_dat_1.txt"); // dalis
    //ifstream fd("IFF06ArnasBurba_L3_dat_2.txt"); // mazai
	ifstream fd("IFF06ArnasBurba_L3_dat_3.txt"); // viskas
    ofstream fr("res.txt");

    BusinessManager businessManagers[300];
    size_t count = 0;

    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);

    while (!fd.eof())
    {
        BusinessManager manager;
        string tempName;
        fd >> tempName;
		strcpy(manager.name, tempName.c_str());
        fd >> manager.workYears;
        fd >> manager.averageProfit;

        businessManagers[count++] = manager;
    }

	fd.close();
    
    //cpu
	Result* result = new Result[count];
    
    //gpu
    BusinessManager* input_managers;
	Result* output_result;
    
    hipError_t errors;

    // nuo kada prasideda blokas
    int Block_start[Blocks_Count];
    // kiek elementu blokui
    int Block_elements[Blocks_Count];
	
    // bloku padalinimas
    int size = count;
    int bloc = Blocks_Count;
    for (int i = 0; i < Blocks_Count; i++)
    {
        Block_elements[i] = size / bloc;
        size -= Block_elements[i];
        bloc--;
        if (i == 0) {
            Block_start[i] = 0;
        }
        else
        {
            Block_start[i] = Block_start[i - 1] + Block_elements[i - 1];
        }
    }

	//perduodam gpu bloku pradzia ir elementu kiekius
    errors = hipMemcpyToSymbol(HIP_SYMBOL(BlockStart), Block_start, sizeof(Block_start), 0, hipMemcpyHostToDevice);
    if (errors != hipSuccess) { return 1; }
    errors = hipMemcpyToSymbol(HIP_SYMBOL(Elements_for_Block), Block_elements, sizeof(Block_elements), 0, hipMemcpyHostToDevice);
    if (errors != hipSuccess) { return 2; }

    // isskiriam vietos 
    errors = hipMalloc((void**)&input_managers, count * sizeof(BusinessManager));
    if (errors != hipSuccess) { return 3; }

    // kopijuojam duomenis
    errors = hipMemcpy(input_managers, businessManagers, count * sizeof(BusinessManager), hipMemcpyHostToDevice);
    if (errors != hipSuccess) { return 4; }

    // isskiriama vietos rezultato masyvui
    errors = hipMalloc((void**)&output_result, count * sizeof(Result));
    if (errors != hipSuccess) { return 5; }

	//paleidziama uzduotis
    Task <<<Blocks_Count, Threads_Count >> > (output_result, input_managers);

    //sulaukiam visu giju pabaigos
    errors = hipDeviceSynchronize();
    if (errors != hipSuccess) { return 6; }

    // Nusikopijuojame rezultatus i cpu/host
    errors = hipMemcpy(result, output_result, count * sizeof(Result), hipMemcpyDeviceToHost);
    if (errors != hipSuccess) { return 7; }

    // atlaisviname atminti
    hipFree(input_managers);
    hipFree(output_result);

    printf("Printing to file\n");

    for (int i = 0; i < Blocks_Count; i++)
    {
        for (int j = 0; j < Block_elements[i]; j++)
        {
            if (!result[Block_start[i] + j].res[0])
            {
                break;
            }

            fr << result[Block_start[i] + j].res << endl;
        }
    }
    
    return 0;
}

